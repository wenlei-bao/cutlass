#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2023 - 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>

#include "cutlass/util/print_error.hpp"
#include "cutlass/util/GPU_Clock.hpp"
#if defined(CUTLASS_ENABLE_CUBLAS) && CUTLASS_ENABLE_CUBLAS != 0
#  include "cutlass/util/cublas_wrappers.hpp"
#endif
#include "cutlass/util/helper_cuda.hpp"

// Extra includes
/// collective builder
// #include "cutlass/util/command_line.h"
// #include "cutlass/kernel_hardware_info.hpp"

// #include "cutlass/cutlass.h"
// #include "cutlass/tensor_ref.h"
// #include "cutlass/epilogue/collective/default_epilogue.hpp"
// #include "cutlass/epilogue/thread/linear_combination.h"
// #include "cutlass/gemm/dispatch_policy.hpp"
// #include "cutlass/gemm/collective/collective_builder.hpp"
// #include "cutlass/epilogue/collective/collective_builder.hpp"
// #include "cutlass/gemm/device/gemm_universal_adapter.h"
// #include "cutlass/gemm/kernel/gemm_universal.hpp"
// #include "cutlass/gemm/kernel/tile_scheduler.hpp"

// #include "cutlass/util/command_line.h"
// #include "cutlass/util/distribution.h"
// #include "cutlass/util/host_tensor.h"
// #include "cutlass/util/packed_stride.hpp"
// #include "cutlass/util/tensor_view_io.h"
// #include "cutlass/util/reference/device/gemm_complex.h"
// #include "cutlass/util/reference/device/tensor_compare.h"
// #include "cutlass/util/reference/device/tensor_fill.h"
//// from example 47
#include <iostream>
#include <string>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm_universal.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/tensor_view_io.h"

#include "helper.h"
// Extra includes end


template <class MShape, class NShape, class KShape,
          class TA, class AStride, class ABlockLayout, class AThreadLayout,
          class TB, class BStride, class BBlockLayout, class BThreadLayout,
          class TC, class CStride, class CBlockLayout, class CThreadLayout,
          class Alpha, class Beta>
__global__ static
__launch_bounds__(decltype(size(CThreadLayout{}))::value)
void
gemm_device(MShape M, NShape N, KShape K,
            TA const* A, AStride dA, ABlockLayout blockA, AThreadLayout tA,
            TB const* B, BStride dB, BBlockLayout blockB, BThreadLayout tB,
            TC      * C, CStride dC, CBlockLayout       , CThreadLayout tC,
            Alpha alpha, Beta beta)
{
  using namespace cute;
  using X = Underscore;

  // Preconditions
  CUTE_STATIC_ASSERT(is_static<ABlockLayout>::value);
  CUTE_STATIC_ASSERT(is_static<BBlockLayout>::value);
  CUTE_STATIC_ASSERT(is_static<CBlockLayout>::value);

  CUTE_STATIC_ASSERT(is_static<AThreadLayout>::value);
  CUTE_STATIC_ASSERT(is_static<BThreadLayout>::value);
  CUTE_STATIC_ASSERT(is_static<CThreadLayout>::value);

  CUTE_STATIC_ASSERT_V(size(tA) == size(tC));
  CUTE_STATIC_ASSERT_V(size(tB) == size(tC));

  //CUTE_STATIC_ASSERT_V(shape<0>(blockA) == shape<0>(blockC));      // BLK_M
  //CUTE_STATIC_ASSERT_V(shape<0>(blockB) == shape<1>(blockC));      // BLK_N
  CUTE_STATIC_ASSERT_V(shape<1>(blockA) == shape<1>(blockB));        // BLK_K

  // Shared memory buffers
  __shared__ TA smemA[cosize_v<ABlockLayout>];
  __shared__ TB smemB[cosize_v<BBlockLayout>];
  auto sA = make_tensor(make_smem_ptr(smemA), blockA);               // (BLK_M,BLK_K)
  auto sB = make_tensor(make_smem_ptr(smemB), blockB);               // (BLK_N,BLK_K)

  // Represent the full tensors
  auto mA = make_tensor(make_gmem_ptr(A), make_shape(M,K), dA);      // (M,K)
  auto mB = make_tensor(make_gmem_ptr(B), make_shape(N,K), dB);      // (N,K)
  auto mC = make_tensor(make_gmem_ptr(C), make_shape(M,N), dC);      // (M,N)

  // Get the appropriate blocks for this thread block --
  // potential for thread block locality
  auto blk_shape = make_shape(size<0>(sA), size<0>(sB), size<1>(sB));// (BLK_M,BLK_N,BLK_K)
  auto blk_coord = make_coord(blockIdx.x, blockIdx.y, _);            // (m,n,k)

  auto gA = local_tile(mA, blk_shape, blk_coord, Step<_1, X,_1>{});  // (BLK_M,BLK_K,k)
  auto gB = local_tile(mB, blk_shape, blk_coord, Step< X,_1,_1>{});  // (BLK_N,BLK_K,k)
  auto gC = local_tile(mC, blk_shape, blk_coord, Step<_1,_1, X>{});  // (BLK_M,BLK_N)

  //
  // Partition the copying of A and B tiles across the threads
  //

  // TUTORIAL: Example of simple partitioning of A|B tiles over tA|tB
  //   Default is a raked partition, but can be changed with Step<X,Y> parameter

  auto tAgA = local_partition(gA, tA, threadIdx.x);                  // (THR_M,THR_K,k)
  auto tAsA = local_partition(sA, tA, threadIdx.x);                  // (THR_M,THR_K)

  auto tBgB = local_partition(gB, tB, threadIdx.x);                  // (THR_N,THR_K,k)
  auto tBsB = local_partition(sB, tB, threadIdx.x);                  // (THR_N,THR_K)

  //
  // Define C accumulators and A/B partitioning
  //

  // TUTORIAL: Example of partitioning via projections of tC

  // Partition sA (M,K) by the rows of tC
  auto tCsA = local_partition(sA, tC, threadIdx.x, Step<_1, X>{});   // (THR_M,BLK_K)
  // Partition sB (N,K) by the cols of tC
  auto tCsB = local_partition(sB, tC, threadIdx.x, Step< X,_1>{});   // (THR_N,BLK_K)
  // Partition gC (M,N) by the tile of tC
  auto tCgC = local_partition(gC, tC, threadIdx.x, Step<_1,_1>{});   // (THR_M,THR_N)

  // Allocate the accumulators -- same size as the projected data
  auto tCrC = make_fragment_like(tCgC);                              // (THR_M,THR_N)

  // Clear the accumulators
  clear(tCrC);

#if 0
  if(thread0()) {
    print("mA\n");
    print(mA.shape()); print("\n"); print(mA.stride());
    print("\n\ngA\n");
    print(gA.shape()); print("\n"); print(gA.stride());
    print("\n\ntAgA\n");
    print(tAgA.shape()); print("\n"); print(tAgA.stride());
    print("\n\nsA\n");
    print(sA.shape()); print("\n"); print(sA.stride());
    print("\n\ntAsA\n");
    print(tAsA.shape()); print("\n"); print(tAsA.stride());
    print("\n\n");
  }
#endif

#if 0
  if(thread0()) {
    print("mB\n");
    print(mB.shape()); print("\n"); print(mB.stride());
    print("\n\ngB\n");
    print(gB.shape()); print("\n"); print(gB.stride());
    print("\n\ntBgB\n");
    print(tBgB.shape()); print("\n"); print(tBgB.stride());
    print("\n\nsB\n");
    print(sB.shape()); print("\n"); print(sB.stride());
    print("\n\ntBsB\n");
    print(tBsB.shape()); print("\n"); print(tBsB.stride());
    print("\n\n");
  }
#endif

#if 0
  if(thread0()) {
    print("mC\n");
    print(mC.shape()); print("\n"); print(mC.stride());
    print("\n\ngC\n");
    print(gC.shape()); print("\n"); print(gC.stride());
    print("\n\ntCsA\n");
    print(tCsA.shape()); print("\n"); print(tCsA.stride());
    print("\n\ntCsB\n");
    print(tCsB.shape()); print("\n"); print(tCsB.stride());
    print("\n\ntCgC\n");
    print(tCgC.shape()); print("\n"); print(tCgC.stride());
    print("\n\ntCrC\n");
    print(tCrC.shape()); print("\n"); print(tCrC.stride());
    print("\n\n");
  }
#endif

#if 1

  // TUTORIAL: Example of a very simple compute loop
  //   Data is read from global to shared memory via the tA|tB partitioning
  //   gemm(.) operates on the shared memory directly via the tC partitioning

  auto k_max = size<2>(tAgA);

  for (int k = 0; k < k_max; ++k)
  {
    // Copy gmem to smem
    copy(tAgA(_,_,k), tAsA);
    copy(tBgB(_,_,k), tBsB);

    // In case copy uses cp.async, make sure that the cp.async
    // instructions are ordered with respect to other cp.async
    // instructions (fence), then wait on all the outstanding copy
    // operations (wait<0>()).  __syncthreads() alone does not do
    // this.
    //
    // NOTE: cp_async_wait<0>() currently issues cp.async.wait_all.
    // This is equivalent to cp.async.commit_group followed by
    // cp.async_wait_group 0.  This should make the first
    // cp_async_fence() (which also issues cp.async.commit_group)
    // redundant.  The tutorial works as-is, so we'll leave the
    // redundant fence in for now and study its removal later.
    cp_async_fence();
    cp_async_wait<0>();

    __syncthreads();

    // Compute gemm on smem
    gemm(tCsA, tCsB, tCrC);

    __syncthreads();
  }

#endif

  //
  // Epilogue
  //

  axpby(alpha, tCrC, beta, tCgC);
}


template <typename TA, typename TB, typename TC,
          typename Alpha, typename Beta>
void
gemm(int m, int n, int k,
     Alpha alpha,
     TA const* A, int ldA,
     TB const* B, int ldB,
     Beta beta,
     TC      * C, int ldC,
     hipStream_t stream = 0)
{
  using namespace cute;

  // Define shapes (dynamic)
  auto M = int(m);
  auto N = int(n);
  auto K = int(k);

  // Define strides (mixed)
  auto dA = make_stride(Int<1>{}, ldA);
  auto dB = make_stride(Int<1>{}, ldB);
  auto dC = make_stride(Int<1>{}, ldC);

  // Define block sizes (static)
  auto bM = Int<128>{};
  auto bN = Int<128>{};
  auto bK = Int<  8>{};

  // Define the block layouts (static)
  auto sA = make_layout(make_shape(bM,bK));
  auto sB = make_layout(make_shape(bN,bK));
  auto sC = make_layout(make_shape(bM,bN));

  // Define the thread layouts (static)
  auto tA = make_layout(make_shape(Int<32>{}, Int< 8>{}));
  auto tB = make_layout(make_shape(Int<32>{}, Int< 8>{}));
  auto tC = make_layout(make_shape(Int<16>{}, Int<16>{}));

  dim3 dimBlock(size(tC));
  dim3 dimGrid(ceil_div(size(M), size(bM)),
               ceil_div(size(N), size(bN)));
  gemm_device
      <<< dimGrid, dimBlock, 0, stream >>>
      (M,  N,  K,
       A, dA, sA, tA,
       B, dB, sB, tB,
       C, dC, sC, tC,
       alpha, beta);
}

#include <cstdlib>
#include <cstdio>
#include <cassert>

void test_gemm(int m, int n, int k)
{
  cute::device_init(0);

  std::cout << "M = " << m << std::endl;
  std::cout << "N = " << n << std::endl;
  std::cout << "K = " << k << std::endl;

  using TA = float;
  using TB = float;
  using TC = float;
  using TI = float;

  thrust::host_vector<TA> h_A(m*k);
  thrust::host_vector<TB> h_B(n*k);
  thrust::host_vector<TC> h_C(m*n);

  for (int j = 0; j < m*k; ++j) h_A[j] = static_cast<TA>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < n*k; ++j) h_B[j] = static_cast<TB>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < m*n; ++j) h_C[j] = static_cast<TC>(-1);

  thrust::device_vector<TA> d_A = h_A;
  thrust::device_vector<TB> d_B = h_B;
  thrust::device_vector<TC> d_C = h_C;

  TI alpha = 1.0;
  TI beta  = 0.0;

  double gflops = (2.0*m*n*k) * 1e-9;

  const int timing_iterations = 100;
  GPU_Clock timer;

#if defined(CUTLASS_ENABLE_CUBLAS) && CUTLASS_ENABLE_CUBLAS != 0
  //
  // cuBLas
  //

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // Run once
  d_C = h_C;
  blam::cublas::gemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                     m, n, k,
                     &alpha,
                     d_A.data().get(), m,
                     d_B.data().get(), n,
                     &beta,
                     d_C.data().get(), m);
  CUTE_CHECK_LAST();

  thrust::host_vector<TC> cublas_result = d_C;

  // Timing iterations
  timer.start();
  for (int i = 0; i < timing_iterations; ++i) {
    blam::cublas::gemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                       m, n, k,
                       &alpha,
                       d_A.data().get(), m,
                       d_B.data().get(), n,
                       &beta,
                       d_C.data().get(), m);
  }
  double cublas_time = timer.seconds() / timing_iterations;
  CUTE_CHECK_LAST();
  printf("CUBLAS_GEMM:   [%6.1f]GFlop/s  (%6.4f)ms\n", gflops / cublas_time, cublas_time*1000);

#else

  std::cout << "Verification by comparison with cuBLAS is disabled, "
    "either because the CMake option CUTLASS_ENABLE_CUBLAS "
    "was explicitly set to OFF, or because CMake could not find cuBLAS.  "
    "If you would like to enable verification with cuBLAS, "
    "please set the CMake option CUTLASS_ENABLE_CUBLAS to ON, "
    "rerun CMake, and recompile this example.\n";

#endif // CUTLASS_ENABLE_CUBLAS

  //
  // CuTe
  //

  // Run once (and check)
  d_C = h_C;
  gemm(m, n, k,
       alpha,
       d_A.data().get(), m,
       d_B.data().get(), n,
       beta,
       d_C.data().get(), m);
  CUTE_CHECK_LAST();
  thrust::host_vector<TC> cute_result = d_C;

  // Timing iterations
  timer.start();
  for (int i = 0; i < timing_iterations; ++i) {
    gemm(m, n, k,
         alpha,
         d_A.data().get(), m,
         d_B.data().get(), n,
         beta,
         d_C.data().get(), m);
  }
  double cute_time = timer.seconds() / timing_iterations;
  CUTE_CHECK_LAST();
  printf("CUTE_GEMM:     [%6.1f]GFlop/s  (%6.4f)ms\n", gflops / cute_time, cute_time*1000);

#if defined(CUTLASS_ENABLE_CUBLAS) && CUTLASS_ENABLE_CUBLAS != 0
  printf("Empirical Perf: %.1f%%\n", (cublas_time / cute_time) * 100);

  auto host_matrix_to_const_column_major_cute_tensor =
    [](const auto& X, int num_rows, int num_cols, int LDX) {
      const auto shape = cute::Shape<int, int>{num_rows, num_cols};
      const auto strides = cute::Stride<int, int>{1, LDX};
      return cute::make_tensor(X.data(), cute::make_layout(shape, strides));
    };

  const auto A_view = host_matrix_to_const_column_major_cute_tensor(h_A, m, k, m);
  // B^T is k x n, so B is n x k.
  const auto B_view = host_matrix_to_const_column_major_cute_tensor(h_B, n, k, n);
  const auto C_computed_view = host_matrix_to_const_column_major_cute_tensor(cute_result, m, n, m);
  const auto C_expected_view = host_matrix_to_const_column_major_cute_tensor(cublas_result, m, n, m);
  print_matrix_multiply_mollified_relative_error("float", A_view, B_view, C_computed_view, C_expected_view);

#endif // CUTLASS_ENABLE_CUBLAS
}



// TODO: use collective builder
/*
// template<
// class MainloopScheduleType = cutlass::gemm::collective::KernelScheduleAuto,
// class EpilogueScheduleType = cutlass::epilogue::collective::EpilogueScheduleAuto,
// class StageCountType = cutlass::gemm::collective::StageCountAuto,
// class TileSchedulerType = cutlass::gemm::PersistentScheduler
// >
// struct GemmRunner {
//   using LayoutA = cutlass::layout::RowMajor;
//   using LayoutB = cutlass::layout::ColumnMajor;
//   using LayoutC = cutlass::layout::ColumnMajor;
//   using LayoutD = cutlass::layout::ColumnMajor;

//   using ElementA = cutlass::half_t;
//   using ElementB = cutlass::half_t;
//   using ElementC = cutlass::half_t;
//   using ElementD = cutlass::half_t;
//   using ElementAccumulator = float;
//   using ElementCompute = float;
//   using ElementScalar = float;

//   // 16B align ony for TMA?
//   static constexpr int AlignmentA = 16 / sizeof(ElementA);
//   static constexpr int AlignmentB = 16 / sizeof(ElementB);
//   static constexpr int AlignmentC = 16 / sizeof(ElementC);
//   static constexpr int AlignmentD = 16 / sizeof(ElementD);

//   // Epilogue collective has sm80?
//   using CollectiveEpilogue = typename cutlass::epilogue::collective::DefaultEpilogue

// }
*/


// A matrix
using ElementA = cutlass::half_t;
using LayoutA = cutlass::layout::RowMajor;
constexpr int AlignmentA = 128 / cutlass::sizeof_bits<ElementA>::value;
// B matrix
using ElementB = cutlass::half_t;
using LayoutB = cutlass::layout::ColumnMajor;
constexpr int AlignmentB = 128 / cutlass::sizeof_bits<ElementB>::value;
// C matrix
using ElementC = cutlass::half_t;
using LayoutC = cutlass::layout::ColumnMajor;
constexpr int AlignmentC = 128 / cutlass::sizeof_bits<ElementC>::value;

// Multiply-accumulate blocking/pipelining details
using ElementAccumulator = float;
using ArchTag = cutlass::arch::Sm80;
using OperatorClass = cutlass::arch::OpClassTensorOp;

using ThreadblockShape = cutlass::gemm::GemmShape<256, 128, 32>;
using WarpShape = cutlass::gemm::GemmShape<64, 64, 32>;
using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;
constexpr int NumStages = 3;

// Epilogue
using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
  ElementC,
  AlignmentC,
  ElementAccumulator,
  ElementAccumulator>;

using DeviceGemmReference = cutlass::reference::device::Gemm<
  ElementA,
  LayoutA,
  ElementB,
  LayoutB,
  ElementC,
  LayoutC,
  ElementAccumulator,
  ElementAccumulator>;
// Matching profiler generated GEMM
// cutlass::arch::OpMultiplyAdd?
// typename cutlass::gemm::kernel::DefaultGemmUniversal
using DeviceGemmTest = cutlass::gemm::device::GemmUniversal<
ElementA, LayoutA,
ElementB, LayoutB,
ElementC, LayoutC,
ElementAccumulator,
OperatorClass,
ArchTag,
ThreadblockShape,
WarpShape,
InstructionShape,
EpilogueOp,
cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
NumStages,
AlignmentA,
AlignmentB,
cutlass::arch::OpMultiplyAdd>;

struct Result
{
  double avg_runtime_ms;
  double gflops;
  cutlass::Status status;
  hipError_t error;
  bool passed;

  Result(
    double avg_runtime_ms = 0,
    double gflops = 0,
    cutlass::Status status = cutlass::Status::kSuccess,
    hipError_t error = hipSuccess)
  :
    avg_runtime_ms(avg_runtime_ms), gflops(gflops), status(status), error(error), passed(true)
  {}

};


/// Command line options parsing
struct Options
{
  std::string               command_name;
  bool                      help;
  cutlass::gemm::GemmCoord  problem_size;
  float                     alpha;
  float                     beta;
  int                       split_k_factor;
  int                       avail_sms;
  bool                      reference_check;
  int                       iterations;

  cutlass::HostTensor<ElementA, LayoutA> tensor_a;
  cutlass::HostTensor<ElementB, LayoutB> tensor_b;
  cutlass::HostTensor<ElementC, LayoutC> tensor_c;
  cutlass::HostTensor<ElementC, LayoutC> tensor_d;
  cutlass::HostTensor<ElementC, LayoutC> tensor_ref_d;

  Options(std::string command_name) :
    command_name(command_name),
    help(false),
    problem_size({512, 6144, 12288}),
    alpha(1.0f),
    beta(0.0f),
    split_k_factor(1),
    avail_sms(-1),              // Number of device SMs to use is unlimited
    reference_check(true),
    iterations(10000)
  {}

  bool valid() const
  {
    return true;
  }

  void parse(int argc, char const **args)
  {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
    }

    cmd.get_cmd_line_argument("m", problem_size.m());
    cmd.get_cmd_line_argument("n", problem_size.n());
    cmd.get_cmd_line_argument("k", problem_size.k());
    cmd.get_cmd_line_argument("alpha", alpha);
    cmd.get_cmd_line_argument("beta", beta);
    cmd.get_cmd_line_argument("split", split_k_factor);
    cmd.get_cmd_line_argument("iterations", iterations);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const
  {
    out
      << "Performs a GEMM computation.\n"
      << "\n"
      << "Options:\n"
      << "\n"
      << "  --help                      If specified, displays this usage statement.\n\n"
      << "  --m=<int>                   GEMM M dimension\n"
      << "  --n=<int>                   GEMM N dimension\n"
      << "  --k=<int>                   GEMM K dimension\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n\n"
      << "  --split=<int>               Split-K factor to emulate\n\n"
      << "  --iterations=<int>          Number of profiling iterations to perform.\n\n";

    out
      << "\n\nExamples:\n\n"
      << "$ " << command_name << " --m=1024 --n=512 --k=1024 --alpha=2 --beta=0.707 \n\n";

    return out;
  }

  /// Compute performance in GFLOP/s
  double gflops(double runtime_s) const
  {
    // Two flops per multiply-add
    return 2.0 * double(problem_size.product()) / double(1.0e9) / runtime_s;
  }
};


typename DeviceGemmTest::Arguments args_from_options(
  const DeviceGemmTest &device_gemm,
  const Options & options,
  cutlass::HostTensor<ElementA, LayoutA> &tensor_a,
  cutlass::HostTensor<ElementB, LayoutB> &tensor_b,
  cutlass::HostTensor<ElementC, LayoutC> &tensor_c,
  cutlass::HostTensor<ElementC, LayoutC> &tensor_d) {
  return typename DeviceGemmTest::Arguments(
    cutlass::gemm::GemmUniversalMode::kGemm,  // universal mode
    options.problem_size,                     // problem_size
    options.split_k_factor,                   // batch count / splitk slices
    {                                         // epilogue parameters
      ElementAccumulator(options.alpha),
      ElementAccumulator(options.beta)
    },
    tensor_a.device_data(),                   // ptr_A
    tensor_b.device_data(),                   // ptr_B
    tensor_c.device_data(),                   // ptr_C
    tensor_d.device_data(),                   // ptr_D
    options.problem_size.mk().product(),      // batch_stride_A
    options.problem_size.nk().product(),      // batch_stride_B
    options.problem_size.mn().product(),      // batch_stride_C
    options.problem_size.mn().product(),      // batch_stride_D
    tensor_a.layout().stride(0),              // stride_a
    tensor_b.layout().stride(0),              // stride_b
    tensor_c.layout().stride(0),              // stride_c
    tensor_d.layout().stride(0));             // stride_d
}

template <typename DeviceGemmT>
Result run(std::string description, Options &options){
  // Display test description
  std::cout << std::endl << description << std::endl;
  // Zero-initialize test output matrix D
  cutlass::reference::host::TensorFill(options.tensor_d.host_view());
  options.tensor_d.sync_device();
  // Instantiate CUTLASS kernel depending on templates
  DeviceGemmT device_gemm;
  // Create a structure of gemm kernel arguments suitable for invoking an instance of DeviceGemmT
  auto arguments = args_from_options(device_gemm, options, options.tensor_a, options.tensor_b, options.tensor_c, options.tensor_d);
  // Using the arguments, query for extra workspace required for matrix multiplication computation
  size_t workspace_size = DeviceGemmT::get_workspace_size(arguments);
  // Allocate workspace memory
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);
  // Check the problem size is supported or not
  CUTLASS_CHECK(device_gemm.can_implement(arguments));
  // Initialize CUTLASS kernel with arguments and workspace pointer
  CUTLASS_CHECK(device_gemm.initialize(arguments, workspace.get()));
  // Correctness / Warmup iteration
  CUTLASS_CHECK(device_gemm());
  // Copy output data from CUTLASS and reference kernel to host for comparison
  options.tensor_d.sync_host();
  // Check if output from CUTLASS kernel and reference kernel are equal or not
  Result result;
  result.passed = cutlass::reference::host::TensorEquals(
    options.tensor_d.host_view(),
    options.tensor_ref_d.host_view());

  std::cout << "  Disposition: " << (result.passed ? "Passed" : "Failed") << std::endl;

  // Run profiling loop
  if (options.iterations > 0)
  {
    GpuTimer timer;
    timer.start();
    for (int iter = 0; iter < options.iterations; ++iter) {
      CUTLASS_CHECK(device_gemm());
    }
    timer.stop();

    // Compute average runtime and GFLOPs.
    float elapsed_ms = timer.elapsed_millis();
    result.avg_runtime_ms = double(elapsed_ms) / double(options.iterations);
    result.gflops = options.gflops(result.avg_runtime_ms / 1000.0);

    std::cout << "  Avg runtime: " << result.avg_runtime_ms << " ms" << std::endl;
    std::cout << "  GFLOPs: " << result.gflops << std::endl;
  }

  if (!result.passed) {
    exit(-1);
  }

  return result;
}




int main(int argc, char const **argv)
{
  // int m = 5120;
  // if (argc >= 2)
  //   sscanf(argv[1], "%d", &m);

  // int n = 5120;
  // if (argc >= 3)
  //   sscanf(argv[2], "%d", &n);

  // int k = 4096;
  // if (argc >= 4)
  //   sscanf(argv[3], "%d", &k);

  // test_gemm(m, n, k);

  // Current device must must have compute capability at least 80
  hipDeviceProp_t props;
  int current_device_id;
  CUDA_CHECK(hipGetDevice(&current_device_id));
  CUDA_CHECK(hipGetDeviceProperties(&props, current_device_id));
  if (!((props.major * 10 + props.minor) >= 80))
  {
    std::cerr << "Ampere Tensor Core operations must be run on a machine with compute capability at least 80."
              << std::endl;

    // Returning zero so this test passes on older Toolkits. Its actions are no-op.
    return 0;
  }
#if 1
  std::cout << "gpu major:" << props.major << std::endl;
#endif

  // Parse options
  Options options("ampere_gemm");
  options.parse(argc, argv);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  std::cout <<
    options.iterations << " timing iterations of " <<
    options.problem_size.m() << " x " <<
    options.problem_size.n() << " x " <<
    options.problem_size.k() << " matrix-matrix multiply" << std::endl;

  if (!options.valid()) {
    std::cerr << "Invalid problem." << std::endl;
    return -1;
  }

  // The KernelHardwareInfo struct holds the number of SMs on the GPU with a given device ID. This
  // information is used by the underlying kernel.
  cutlass::KernelHardwareInfo hw_info;

  // Change device_id to another value if you are running on a machine with multiple GPUs and wish
  // to use a GPU other than that with device ID 0.
  hw_info.device_id = 0;
  hw_info.sm_count = cutlass::KernelHardwareInfo::query_device_multiprocessor_count(hw_info.device_id);

  #if 1
  std::cout << hw_info.sm_count << std::endl;
  #endif

  // Build up gemm and run

  // Initialize tensors using CUTLASS helper functions
  options.tensor_a.resize(options.problem_size.mk());       // <- Create matrix A with dimensions M x K
  options.tensor_b.resize(options.problem_size.kn());       // <- Create matrix B with dimensions K x N
  options.tensor_c.resize(options.problem_size.mn());       // <- Create matrix C with dimensions M x N
  options.tensor_d.resize(options.problem_size.mn());       // <- Create matrix D with dimensions M x N used to store output from CUTLASS kernel
  options.tensor_ref_d.resize(options.problem_size.mn());   // <- Create matrix D with dimensions M x N used to store output from reference kernel

  // Fill matrix A on host with uniform-random data [-2, 2]
  cutlass::reference::host::TensorFillRandomUniform(
      options.tensor_a.host_view(),
      1,
      ElementA(2),
      ElementA(-2),
      0);

  // Fill matrix B on host with uniform-random data [-2, 2]
  cutlass::reference::host::TensorFillRandomUniform(
      options.tensor_b.host_view(),
      1,
      ElementB(2),
      ElementB(-2),
      0);

  // Fill matrix C on host with uniform-random data [-2, 2]
  cutlass::reference::host::TensorFillRandomUniform(
      options.tensor_c.host_view(),
      1,
      ElementC(2),
      ElementC(-2),
      0);

  // Copy data from host to GPU
  options.tensor_a.sync_device();
  options.tensor_b.sync_device();
  options.tensor_c.sync_device();
  // Zero-initialize reference output matrix D
  cutlass::reference::host::TensorFill(options.tensor_ref_d.host_view());
  options.tensor_ref_d.sync_device();

  // Create instantiation for device reference gemm kernel
  DeviceGemmReference gemm_reference;
  // Launch device reference gemm kernel
  gemm_reference(
    options.problem_size,
    ElementAccumulator(options.alpha),
    options.tensor_a.device_ref(),
    options.tensor_b.device_ref(),
    ElementAccumulator(options.beta),
    options.tensor_c.device_ref(),
    options.tensor_ref_d.device_ref());
  // Wait for kernels to finish
  CUDA_CHECK(hipDeviceSynchronize());
  // Copy output data from reference kernel to host for comparison
  options.tensor_ref_d.sync_host();

  Result gemm_test_res = run<DeviceGemmTest>("Device gemm from profiler", options);
  printf("Done\n");

  return 0;
}
